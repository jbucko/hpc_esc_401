
#include <hip/hip_runtime.h>
#include <stdio.h> 

#include <sys/time.h>

double getTime(void) {
    struct timeval tv;
    struct timezone tz;
    gettimeofday(&tv, &tz);
    return tv.tv_sec + 1e-6*(double)tv.tv_usec;
}

#define NBIN 1000000000 // Number of bins
// #define NUM_BLOCK 500 // Number of thread blocks (max 2^32-1)
// #define NUM_THREAD 64 // Number of threads per block (max 1024)



__global__ void cal_pi(
    double *sum, int nbin, double step) {
    int i;
    double x;
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    for (i=idx; i< nbin; i+= blockDim.x*gridDim.x) {
        x = (i+0.5)*step;
        sum[idx] += 4.0/(1.0+x*x);
    }
}




int main(void) {
    int  blocks[8] = {60,120,180,240,300,360,430,600}; //here we define an array of blocks
    int  threads[10] = {16,32,48,64,80,96,112,128,144,160}; //here we define an array of threads
    int NUM_BLOCK,NUM_THREAD;

    for (int i=0;i<8;i++) for (int j=0;j<10;j++){ //we loop over all the combinations in blocks and threads arrays
        NUM_BLOCK = blocks[i];
        NUM_THREAD = threads[j];
        
        dim3 dimGrid(NUM_BLOCK,1,1); // Grid dimensions
        dim3 dimBlock(NUM_THREAD,1,1); // Block dimensions
        double *sumHost, *sumDev; // Pointer to host & device arrays
        double pi = 0;
        int tid;
        double step = 1.0/NBIN; // Step size
        size_t size = NUM_BLOCK*NUM_THREAD*sizeof(double);
        sumHost = (double *)malloc(size); // array on host
        hipMalloc((void **) &sumDev, size);// array on GPU
        hipMemset(sumDev, 0, size); // Zero results array

        double start, end;
        start = getTime();
        cal_pi <<<dimGrid, dimBlock>>>(sumDev,NBIN,step); // call CUDA kernel
        // Retrieve result from device and store it in host array
        hipMemcpy(sumHost, sumDev, size, hipMemcpyDeviceToHost);
        for(tid=0; tid<NUM_THREAD*NUM_BLOCK; tid++)
        pi += sumHost[tid];
        pi *= step;
        // Cleanup
        end=getTime();
        double delta = end - start;
        printf("NUM_BLOCK: %d, NUM_THREAD: %d -> PI = %.16g computed in %.4g seconds\n", NUM_BLOCK, NUM_THREAD, pi, delta);
        free(sumHost);
        hipFree(sumDev);
    }
}
